#include "hip/hip_runtime.h"
/*
* This sample implements a separable convolution 
* of a 2D image with an arbitrary filter.
*/

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
const unsigned int filter_radius=16;



hipError_t code;

#define CUDA_ERROR_CHECK(n) \
   code = hipGetLastError(); \
   if ( code != hipSuccess ) {\
    printf("**** Error at num %d hipGetLastError().*********\n", n ); \
    printf("Type of error: %s\n", hipGetErrorString( code )); \
   }

#define FILTER_LENGTH 	(2 * filter_radius + 1)


#define ABS(val)  	((val)<0.0 ? (-(val)) : (val))
#define accuracy  	0.00005 

 

__constant__  __device__ float d_Filter[FILTER_LENGTH];

////////////////////////////////////////////////////////////////////////////////
// Reference row convolution filter
////////////////////////////////////////////////////////////////////////////////
void convolutionRowCPU(float *h_Dst, float *h_Src, float *h_Filter, 
                       int imageW, int imageH, int filterR) {

  int x, y, k;
                      
  for (y = 0; y < imageH; y++) {
    for (x = 0; x < imageW; x++) {
      float sum = 0;

      for (k = -filterR; k <= filterR; k++) {
        int d = x + k;

        if (d >= 0 && d < imageW) {
          sum += h_Src[y * imageW + d] * h_Filter[filterR - k];
        }     

        h_Dst[y * imageW + x] = sum;
      }
    }
  }
        
}




////////////////////////////////////////////////////////////////////////////////
// Reference column convolution filter
////////////////////////////////////////////////////////////////////////////////
void convolutionColumnCPU(	
          float *h_Dst, 
          float *h_Src, 
          float *h_Filter,
          int imageW, 
          int imageH, 
          int filterR) 			{

  int x, y, k;
  


  for (y = 0; y < imageH; y++) {
    for (x = 0; x < imageW; x++) {
      float sum = 0;

      for (k = -filterR; k <= filterR; k++) {
        int d = y + k;

        if (d >= 0 && d < imageH) {
          sum += h_Src[d * imageW + x] * h_Filter[filterR - k];
        }   
        h_Dst[y * imageW + x] = sum;
      }
    }
  }    
}


/*
 * GPU convolution Rows
 */ 


__global__ void convolutionRowGPU(
          float *d_Dst, 
          float *d_Src, 
          /*float *d_Filter,*/
          int imageW, 
          int imageH, 
          int filterR)                           
{

	__shared__ float sh_Src[32][32];
      
	int col = threadIdx.x + blockDim.x * blockIdx.x;
	int row = threadIdx.y + blockDim.y * blockIdx.y;
        int index = row*imageW+col;

/*
	if (index == 4095)  {
   
   	  printf("EKTUPWTHIKEEEEEE...\nindex=%d, blockDim.y=%d, blockIdx.y=%d, threadIDx.y=%d\n", index, blockDim.y, blockIdx.y, threadIdx.y);
	   printf("blockDim.x=%d, blockIdx.x=%d, threadIDx.x=%d\n", blockDim.x, blockIdx.x, threadIdx.x);
	  printf("gridDim.x=%d, gridDim.y=%d\n", gridDim.x, gridDim.y);
        }
*/
	sh_Src[threadIdx.y][threadIdx.x] = d_Src[index];
	


	__syncthreads();	  
           
      
        int  k;          
 
      float sum = 0;
      for (k = -filterR; k <= filterR; k++) {

        int d = col + k;
	
        if (d >= 0 && d < imageW) {
          sum += sh_Src[threadIdx.y][ threadIdx.x + k ] * d_Filter[filterR - k];
        }     
	
        d_Dst[index] = sum;
      }  	
}

//sh_Src[threadIdx.y][ threadIdx.x + k ]
//sh_Src[ threadIdx.y + k ][threadIdx.x] 
/*
 * GPU convolution Columns
 */
__global__ void convolutionColumnGPU(
          float *d_Dst, 
          float *d_Src, 
          /*float *d_Filter,*/
          int imageW, 
          int imageH, 
          int filterR)                           
{		
	__shared__ float sh_Src[32][32];

	

	int col = threadIdx.x + blockDim.x * blockIdx.x;
	int row = threadIdx.y + blockDim.y * blockIdx.y;

	
        int index = row*imageW+col;	

	sh_Src[threadIdx.y][threadIdx.x] = d_Src[index];

	__syncthreads();         
	 

    
          int  k;          
       


      float sum = 0;

      for (k = -filterR; k <= filterR; k++) {
        int d = row + k;
	
        if (d >= 0 && d < imageH) {//(y+k)*WIDTH + y = y *WIDTH + k*WIDTH + y = row - 1 + col
          sum += sh_Src[ threadIdx.y + k ][threadIdx.x] * d_Filter[filterR - k];
        }   
        d_Dst[index] = sum;
      }
   
	/* 
      for (y = 0; y < imageH; y++) {//x = cols
    for (x = 0; x < imageW; x++) {y = rows
      float sum = 0;// 

      for (k = -filterR; k <= filterR; k++) {
        int d = y + k;

        if (d >= 0 && d < imageH) {
          sum += h_Src[d * imageW + x] * h_Filter[filterR - k];
        }   
        h_Dst[y * imageW + x] = sum;
      }*/
}


////////////////////////////////////////////////////////////////////////////////
// Main program
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv) {
    
    float *h_Filter, *h_Input, *h_Buffer, *h_OutputCPU, *h_OutputGPU, *d_Input,
*d_Output_GPU, *d_Buffer  /*,*d_Filter*/;
    
    int pointsThatDiffer = 0;
    int imageW;
    int imageH;
    unsigned int i;
    

    	
    // Ta imageW, imageH ta dinei o xrhsths kai thewroume oti einai isa,
    // dhladh imageW = imageH = N, opou to N to dinei o xrhsths.
    // Gia aplothta thewroume tetragwnikes eikones.  
   if (argc < 2)  {
      printf("Few arguments. Run as ./<name>  <image_size>,where <image_size> should be a power of two and greater than 33\n");
      return -1;
   } 

   if ( strlen(argv[1]) == 0 ) {
      printf("Error at argv[1]. Please give the size of image as 1st argument(e.g. ./exe 100 5\n"); 
      return -1;
    }
    imageW = atoi(argv[1]);
    imageH = imageW;

 
    printf("Image Width x Height = %i x %i\n\n", imageW, imageH);
    printf("Allocating and initializing host arrays...\n");
    // Tha htan kalh idea na elegxete kai to apotelesma twn malloc...
    h_Filter    = (float *)malloc(FILTER_LENGTH * sizeof(float));
    h_Input     = (float *)malloc(imageW * imageH * sizeof(float) );
    h_Buffer    = (float *)malloc(imageW * imageH * sizeof(float));
    h_OutputCPU = (float *)malloc(imageW * imageH * sizeof(float));
    
    /// *** EDITED  ***//
    hipMalloc( (void **)&d_Input, imageW * imageH * sizeof(float) );
    //hipMalloc( (void **)&d_Filter, FILTER_LENGTH * sizeof(float) );
    hipMalloc( (void **)&d_Output_GPU,  imageW * imageH * sizeof(float) );
    hipMalloc( (void **)&d_Buffer,  imageW * imageH * sizeof(float) );
    
    h_OutputGPU = (float *)malloc(imageW * imageH * sizeof(float));
    if ( h_Filter == NULL || h_Input == NULL ||  h_Buffer == NULL ||
h_OutputCPU==NULL || h_OutputGPU == NULL) {
      printf("Error allocating host or device\n"); 
    } 

   
   /*
    * tsekare an uparxoun sfalmata
    */
    
    // to 'h_Filter' apotelei to filtro me to opoio ginetai to convolution kai
    // arxikopoieitai tuxaia. To 'h_Input' einai h eikona panw sthn opoia ginetai
    // to convolution kai arxikopoieitai kai auth tuxaia.

    srand(200);

    for (i = 0; i < FILTER_LENGTH; i++) {
        h_Filter[i] = (float)(rand() % 16);
    }

    for (i = 0; i < (unsigned int)imageW * imageH; i++) {
        h_Input[i] = (float)rand() / ((float)RAND_MAX / 255) + (float)rand() / (float)RAND_MAX;
    }

    
    hipMemcpy(d_Input,h_Input,imageW*imageH*sizeof(float),hipMemcpyHostToDevice);
    CUDA_ERROR_CHECK(1);
    
    
   
    code = hipMemcpyToSymbol(HIP_SYMBOL(
       d_Filter),
       h_Filter,
       FILTER_LENGTH*sizeof( float )
    ); if (code != hipSuccess) printf("Error copying from host Memory to Constant Memory!\n");
     

    CUDA_ERROR_CHECK(2);

    // To parakatw einai to kommati pou ekteleitai sthn CPU kai me vash auto prepei na ginei h sugrish me thn GPU.
    printf("CPU computation...\n");

    convolutionRowCPU(h_Buffer, h_Input, h_Filter, imageW, imageH, filter_radius); // convolution kata grammes
    convolutionColumnCPU(h_OutputCPU, h_Buffer, h_Filter, imageW, imageH, filter_radius);//convolution kata sthles    


  /*
   * calculate threads per block
   */


  dim3 threadsPerBlock(32,32);
  dim3 numBlocks(imageW/threadsPerBlock.x, imageH/threadsPerBlock.y);


 

    convolutionRowGPU<<<numBlocks , threadsPerBlock>>>(d_Buffer,
d_Input/*,d_Filter*/, imageH, imageW, filter_radius);
    
    hipDeviceSynchronize();//barrier of host     
    CUDA_ERROR_CHECK(3);
         
    convolutionColumnGPU<<<numBlocks, threadsPerBlock>>>(d_Output_GPU, d_Buffer,
 /*d_Filter,*/ imageH, imageW, filter_radius);
    hipDeviceSynchronize();//barrier of host
    CUDA_ERROR_CHECK(4);





    //return data to host by copying the from global memory to host memory
    hipMemcpy(h_OutputGPU, d_Output_GPU, imageW * imageH * sizeof(float),hipMemcpyDeviceToHost);
    CUDA_ERROR_CHECK(5);

    //now compare host results VS device results. Is GPU same as CPU?!
    for (i = 0; i < (unsigned int)imageW * imageH; i++) {
        if(ABS(h_OutputCPU[i] - h_OutputGPU[i]) > accuracy){
          pointsThatDiffer = 1;
          printf("The difference between the %dnth element is larger than accuracy. \n CPU: %g GPU %g differece: %.15g \nNow exiting..\n", i,h_OutputCPU[i] ,h_OutputGPU[i], ABS(h_OutputGPU[i] - h_OutputCPU[i])  );
	  break;
        }
     }
     if (pointsThatDiffer == 0)
       printf("******************** Correct: GPU output is the same as CPU output *************\n");
     else
       printf("******************** Error: GPU output differs from CPU output!!!  *************\n");

    
    // free all the allocated memory
    free(h_OutputCPU); hipFree(d_Output_GPU);
    free(h_Buffer); hipFree(d_Buffer);
    free(h_Input); hipFree(d_Input);
    free(h_Filter); //hipFree(d_Filter);

    
    // Do a device reset just in case... Bgalte to sxolio otan ylopoihsete CUDA
    hipDeviceReset();
    CUDA_ERROR_CHECK(6);

    return 0;
}
